#include <assert.h>
#include <iostream>
#include <memory>
#include <thread>

#include <hip/hip_runtime.h>
#include <unistd.h>
#include "../check.h"
#include "comm.h"
// #include "cudnn_conv.h"
#include "cutlass_matmul.h"

hipError_t CutlassSgemmNN(int M, int N, int K, float alpha, float const* A,
                           int lda, float const* B, int ldb, float beta,
                           float* C, int ldc,
                           hipStream_t s = (hipStream_t)0) {
    // Define type definition for single-precision CUTLASS GEMM with
    // column-major input matrices and 128x128x8 threadblock tile size (chosen
    // by default).
    //
    // To keep the interface manageable, several helpers are defined for
    // plausible compositions including the following example for
    // single-precision GEMM. Typical values are used as default template
    // arguments. See `cutlass/gemm/device/default_gemm_configuration.h` for
    // more details.
    //
    // To view the full gemm device API interface, see
    // `cutlass/gemm/device/gemm.h`

    using ColumnMajor = cutlass::layout::ColumnMajor;

    using ShapeMMAThreadBlock =
            cutlass::gemm::GemmShape<32, 32, 32>;  // <- threadblock tile M =
                                                    // 128, N = 128, K = 32
    // This code section describes tile size a warp will compute
    using ShapeMMAWarp =
            cutlass::gemm::GemmShape<32, 32, 32>;  // <- warp tile M = 64, N = 64, K = 32
    // This code section describes the size of MMA op
    using ShapeMMAOp = cutlass::gemm::GemmShape<1, 1, 1>;  // <- MMA Op tile M =
                                                           // 8, N = 8, K = 4
    using CutlassGemm =
            cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                        ColumnMajor,  // Layout of A matrix
                                        float,        // Data-type of B matrix
                                        ColumnMajor,  // Layout of B matrix
                                        float,        // Data-type of C matrix
                                        ColumnMajor,  // Layout of C matrix
                                        float,
                                        cutlass::arch::OpClassSimt,
                                        cutlass::arch::Sm70,
                                        ShapeMMAThreadBlock,
                                        ShapeMMAWarp,
                                        ShapeMMAOp>;

    cutlass::device_memory::allocation<uint8_t> workspace(0);
    // Define a CUTLASS GEMM type
    CutlassGemm gemm_operator;

    // Construct the CUTLASS GEMM arguments object.
    //
    // One of CUTLASS's design patterns is to define gemm argument objects that
    // are constructible in host code and passed to kernels by value. These may
    // include pointers, strides, scalars, and other arguments needed by Gemm
    // and its components.
    //
    // The benefits of this pattern are (1.) a structured, composable strategy
    // for passing host-constructible arguments to kernels and (2.) minimized
    // initialization overhead on kernel entry.
    //
    CutlassGemm::Arguments args(
            {M, N, K},  // Gemm Problem dimensions
            {A, lda},   // Tensor-ref for source matrix A
            {B, ldb},   // Tensor-ref for source matrix B
            {C, ldc},   // Tensor-ref for source matrix C
            {C, ldc},   // Tensor-ref for destination matrix D (may be different
                        // memory than source C matrix)
            {alpha, beta});  // Scalars used in the Epilogue

    //
    // Launch the CUTLASS GEMM kernel.
    //
    cutlass::Status status = gemm_operator.initialize(args, workspace.get(), s);
    CUTLASS_CHECK(status);

    status = gemm_operator(s);
    CUTLASS_CHECK(status);

    // Return success, if no errors were encountered.
    return hipSuccess;
}

void run_nccl(int rank, std::unique_ptr<Comm>& comm, size_t nbytes) {
    CUDACHECK(hipSetDevice(rank));
    size_t N = nbytes;
    hipEvent_t start, stop;
    CUDACHECK(hipEventCreate(&start));
    CUDACHECK(hipEventCreate(&stop));

    void* data = nullptr;
    CUDACHECK(hipMalloc(&data, N));
    CUDACHECK(hipMemsetAsync(data, 0, N, comm->getStream()));
    int times = 100;
    int effective = 100;
    for (int i = 0; i < times + 10; i++) {
        if (i == 10) {
            CUDACHECK(hipEventRecord(start, comm->getStream()));
        }
        comm->allReduce(data, N / 4, ncclFloat32, ncclSum);
        if (i == effective + 9) {
            CUDACHECK(hipEventRecord(stop, comm->getStream()));
        }
    }
    CUDACHECK(hipEventSynchronize(stop));
    float sum = 0.f;
    CUDACHECK(hipEventElapsedTime(&sum, start, stop));

    CUDACHECK(hipFree(data));
    CUDACHECK(hipEventDestroy(start));
    CUDACHECK(hipEventDestroy(stop));

    std::cout << "single nccl " << N
              << " bytes, average time: " << sum / effective << "ms"
              << std::endl;
}

void run_nccl_nccl(int rank, std::unique_ptr<Comm>& comm,
                   std::unique_ptr<Comm>& comm1, size_t nbytes) {
    CUDACHECK(hipSetDevice(rank));
    hipEvent_t start, start1;
    CUDACHECK(hipEventCreate(&start));
    CUDACHECK(hipEventCreate(&start1));

    void* data = nullptr;
    CUDACHECK(hipMalloc(&data, nbytes));
    CUDACHECK(hipMemsetAsync(data, 0, nbytes, comm->getStream()));
    void* data1 = nullptr;
    CUDACHECK(hipMalloc(&data1, nbytes));
    CUDACHECK(hipMemsetAsync(data1, 0, nbytes, comm1->getStream()));

    int times = 1000;
    CUDACHECK(hipEventRecord(start, comm->getStream()));
    CUDACHECK(hipEventRecord(start1, comm1->getStream()));
    CUDACHECK(hipStreamWaitEvent(comm->getStream(), start1, 0));
    CUDACHECK(hipStreamWaitEvent(comm1->getStream(), start1, 0));
    for (int i = 0; i < times + 10; i++) {
        comm->allReduce(data, nbytes, ncclInt8, ncclSum);
        comm1->allReduce(data, nbytes, ncclInt8, ncclSum);
    }
    hipStreamSynchronize(comm->getStream());
    hipStreamSynchronize(comm1->getStream());
}

// void run_cudnn(int rank, int argnum = 4) {
//     CUDACHECK(hipSetDevice(rank));
//     auto arg = arg_list[argnum];
//     int batch_size = arg[0];
//     int C = arg[1];
//     int H = arg[2];
//     int W = arg[3];
//     int kernel_size = arg[4];
//     int K = arg[5];
//     int stride = arg[6];
//     int padding = arg[7];
//     Conv conv(C, K, H, W, batch_size, kernel_size, stride, padding);

//     hipEvent_t start, stop;
//     CUDACHECK(hipEventCreate(&start));
//     CUDACHECK(hipEventCreate(&stop));
//     int times = 100;
//     int effective = 100;
//     for (int i = 0; i < times + 10; i++) {
//         if (i == 10) {
//             CUDACHECK(hipEventRecord(start, conv.getStream()));
//         }
//         conv.forward();
//         if (i == effective + 9) {
//             CUDACHECK(hipEventRecord(stop, conv.getStream()));
//         }
//     }
//     CUDACHECK(hipEventSynchronize(stop));
//     float sum = 0.0;
//     CUDACHECK(hipEventElapsedTime(&sum, start, stop));

//     std::cout << "single cudnn: (" << arg[0] << "," << arg[1] << "," <<
//     arg[2]
//               << "," << arg[3] << "," << arg[4] << "," << arg[5] << ","
//               << arg[6] << "," << arg[7] << ")"
//               << " average time " << sum / effective << "ms" << std::endl;
//     CUDACHECK(hipEventDestroy(start));
//     CUDACHECK(hipEventDestroy(stop));
// }

// void run_cudnn_nccl(int rank, std::unique_ptr<Comm>& comm, size_t nbytes,
//                     int argnum = 4) {
//     CUDACHECK(hipSetDevice(rank));
//     auto arg = arg_list[argnum];
//     int batch_size = arg[0];
//     int C = arg[1];
//     int H = arg[2];
//     int W = arg[3];
//     int kernel_size = arg[4];
//     int K = arg[5];
//     int stride = arg[6];
//     int padding = arg[7];
//     Conv conv(C, K, H, W, batch_size, kernel_size, stride, padding);

//     // nccl:
//     // int N = 500000000;
//     size_t N = nbytes;
//     void* data = nullptr;
//     CUDACHECK(hipMalloc(&data, N));
//     CUDACHECK(hipMemsetAsync(data, 0, N, comm->getStream()));

//     hipEvent_t start, stop, start1, stop1;
//     CUDACHECK(hipEventCreate(&start));
//     CUDACHECK(hipEventCreate(&stop));
//     CUDACHECK(hipEventCreate(&start1));
//     CUDACHECK(hipEventCreate(&stop1));
//     CUDACHECK(hipEventRecord(start, conv.getStream()));
//     CUDACHECK(hipEventRecord(start1, comm->getStream()));
//     CUDACHECK(hipStreamWaitEvent(comm->getStream(), start, 0));
//     CUDACHECK(hipStreamWaitEvent(conv.getStream(), start1, 0));
//     int times = 1000;
//     int effective = 100;
//     for (int i = 0; i < times + 10; i++) {
//         if (i == 10) {
//             CUDACHECK(hipEventRecord(start, conv.getStream()));
//             CUDACHECK(hipEventRecord(start1, comm->getStream()));
//         }
//         conv.forward();
//         comm->allReduce(data, N / 4, ncclFloat32, ncclSum);
//         if (i == effective + 9) {
//             CUDACHECK(hipEventRecord(stop, conv.getStream()));
//             CUDACHECK(hipEventRecord(stop1, comm->getStream()));
//         }
//     }
//     CUDACHECK(hipEventSynchronize(stop));
//     CUDACHECK(hipEventSynchronize(stop1));
//     float sum = 0.0, sum1 = 0.0;
//     CUDACHECK(hipEventElapsedTime(&sum, start, stop));
//     CUDACHECK(hipEventElapsedTime(&sum1, start1, stop1));

//     CUDACHECK(hipEventDestroy(start));
//     CUDACHECK(hipEventDestroy(stop));
//     CUDACHECK(hipEventDestroy(start1));
//     CUDACHECK(hipEventDestroy(stop1));
//     std::cout << "overlap nccl time: " << sum1 / effective << "ms" <<
//     std::endl; std::cout << "overlap cudnn time: " << sum / effective << "ms"
//     << std::endl;
// }

void run_cutlass(int rank, int M = 1024, int N = 1024, int K = 1024) {
    CUDACHECK(hipSetDevice(rank));
    hipStream_t stream = (hipStream_t)0;
    CUDACHECK(hipStreamCreate(&stream));
    float alpha = 1.0, beta = 0.0;
    int lda = M;
    int ldb = K;
    int ldc = M;
    float* A;
    float* B;
    float* C_cutlass;
    AllocateMatrix(&A, lda, M, K, 0, stream);
    AllocateMatrix(&B, ldb, K, N, 17, stream);
    AllocateMatrix(&C_cutlass, ldc, M, N, 101, stream);

    hipEvent_t start, stop;
    CUDACHECK(hipEventCreate(&start));
    CUDACHECK(hipEventCreate(&stop));
    int times = 100;
    int effective = 100;
    for (int i = 0; i < times + 10; i++) {
        if (i == 10) {
            CUDACHECK(hipEventRecord(start, stream));
        }
        CutlassSgemmNN(M, N, K, alpha, A, lda, B, ldb, beta, C_cutlass, ldc,
                       stream);
        if (i == effective + 9) {
            CUDACHECK(hipEventRecord(stop, stream));
        }
    }
    CUDACHECK(hipEventSynchronize(stop));
    float sum = 0.0;
    CUDACHECK(hipEventElapsedTime(&sum, start, stop));

    std::cout << "single cutlass: "
              << " average time " << sum / effective << "ms" << std::endl;
    CUDACHECK(hipEventDestroy(start));
    CUDACHECK(hipEventDestroy(stop));
}

void run_cutlass_nccl(int rank, std::unique_ptr<Comm>& comm, size_t nbytes,
                      int M = 1024, int N = 1024, int K = 1024) {
    CUDACHECK(hipSetDevice(rank));
    hipStream_t stream = (hipStream_t)0;
    CUDACHECK(hipStreamCreate(&stream));
    float alpha = 1.0, beta = 0.0;
    int lda = M;
    int ldb = K;
    int ldc = M;
    float* A;
    float* B;
    float* C_cutlass;
    AllocateMatrix(&A, lda, M, K, 0, stream);
    AllocateMatrix(&B, ldb, K, N, 17, stream);
    AllocateMatrix(&C_cutlass, ldc, M, N, 101, stream);

    // nccl:
    void* data = nullptr;
    CUDACHECK(hipMalloc(&data, nbytes));
    CUDACHECK(hipMemsetAsync(data, 0, nbytes, comm->getStream()));

    hipEvent_t start, stop, start1, stop1;
    CUDACHECK(hipEventCreate(&start));
    CUDACHECK(hipEventCreate(&stop));
    CUDACHECK(hipEventCreate(&start1));
    CUDACHECK(hipEventCreate(&stop1));
    CUDACHECK(hipEventRecord(start, stream));
    CUDACHECK(hipEventRecord(start1, comm->getStream()));
    CUDACHECK(hipStreamWaitEvent(comm->getStream(), start, 0));
    CUDACHECK(hipStreamWaitEvent(stream, start1, 0));
    int times = 1000;
    int effective = 100;
    for (int i = 0; i < times + 10; i++) {
        if (i == 10) {
            CUDACHECK(hipEventRecord(start, stream));
            CUDACHECK(hipEventRecord(start1, comm->getStream()));
        }
        CutlassSgemmNN(M, N, K, alpha, A, lda, B, ldb, beta, C_cutlass, ldc,
                       stream);
        comm->allReduce(data, nbytes / 4, ncclFloat32, ncclSum);
        if (i == effective + 9) {
            CUDACHECK(hipEventRecord(stop, stream));
            CUDACHECK(hipEventRecord(stop1, comm->getStream()));
        }
    }
    CUDACHECK(hipEventSynchronize(stop));
    CUDACHECK(hipEventSynchronize(stop1));
    float sum = 0.0, sum1 = 0.0;
    CUDACHECK(hipEventElapsedTime(&sum, start, stop));
    CUDACHECK(hipEventElapsedTime(&sum1, start1, stop1));

    CUDACHECK(hipEventDestroy(start));
    CUDACHECK(hipEventDestroy(stop));
    CUDACHECK(hipEventDestroy(start1));
    CUDACHECK(hipEventDestroy(stop1));
    std::cout << "overlap nccl time: " << sum1 / effective << "ms" << std::endl;
    std::cout << "overlap cutlass gemm time: " << sum / effective << "ms"
              << std::endl;
}

// ./test <ip> <port> <rank>
int main(int argc, char* argv[]) {
    //run_cutlass(0);
    //hipDeviceSynchronize();
    assert(argc == 4);

    int nrank = 2;
    int rank = atoi(argv[3]);
    const char* ip = argv[1];
    unsigned short port = (unsigned short)atoi(argv[2]);

    std::unique_ptr<Comm> comm =
            std::make_unique<Comm>(nrank, rank, rank, ip, port);

    size_t nccl_nbytes = 5000000;
    run_nccl(rank, comm, nccl_nbytes);
    hipDeviceSynchronize();
    std::cout << std::endl;

    // run_cudnn(rank);
    run_cutlass(rank);
    hipDeviceSynchronize();
    std::cout << std::endl;

    // run_cudnn_nccl(rank, comm, nccl_nbytes);
    run_cutlass_nccl(rank, comm, nccl_nbytes);
    hipDeviceSynchronize();
    return 0;
}
