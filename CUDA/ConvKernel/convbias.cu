#include "hip/hip_runtime.h"
#include "cudnn_convbias.h"
#include "cudnn_conv.h"
#include "../helper.h"

#include "cutlass/cutlass.h"
#include "cutlass/layout/layout.h"
#include "cutlass/conv/kernel/default_conv2d_fprop.h"

#include "cutlass/tensor_coord.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_fill.h"

using ElementInputA = float;             // Data type of elements in input tensor
using ElementInputB = float;             // Data type of elements in input tensor
using ElementOutput = float;             // Data type of elements in output tensor
using ElementAccumulator = float;        // Data type of accumulator
using ElementComputeEpilogue = float;    // Data type of epilogue computation (alpha, beta)

using LayoutInputA = cutlass::layout::TensorNHWC;
using LayoutInputB = cutlass::layout::TensorNHWC;
using LayoutOutput = cutlass::layout::TensorNHWC;

int main() {
    int N = 5;
    int iC = 69; // C
    int iH = 31;
    int iW = 95;
    int oC = 64; // K
    int kH = 3;  // R
    int kW = 3;  // S
    int strideH = 1;
    int strideW = 1;
    int paddingH = 1;  // int paddingH0 = 1; int paddingH1 = 1;
    int paddingW = 1;  // int paddingW0 = 1; int paddingW1 = 1;
    int oH = (iH + 2 * paddingH - kH) / strideH + 1;
    int oW = (iW + 2 * paddingW - kW) / strideW + 1;

    cutlass::Tensor4DCoord input_size(N, iH, iW, iC);
    cutlass::Tensor4DCoord filter_size(oC, kH, kW, iC);
    cutlass::Tensor4DCoord bias_size(1, 1, 1, oC);
    cutlass::Tensor4DCoord output_size(N, oH, oW, oC);
    cutlass::HostTensor<ElementInputA, LayoutInputA> input(input_size);
    cutlass::HostTensor<ElementInputB, LayoutInputB> filter(filter_size);
    cutlass::HostTensor<ElementOutput, LayoutOutput> bias(bias_size);
    cutlass::HostTensor<ElementOutput, LayoutOutput> output(output_size);
    cutlass::HostTensor<ElementOutput, LayoutOutput> output_ref(output_size);
    //cutlass::reference::host::TensorFillRandomUniform(input.host_view(), 1, ElementInputA(1), ElementInputA(-1), 0);
    //cutlass::reference::host::TensorFillRandomUniform(filter.host_view(), 1, ElementInputB(1), ElementInputB(-1), 0);
    //cutlass::reference::host::TensorFillRandomUniform(bias.host_view(), 1, ElementOutput(1), ElementOutput(-1), 0);
    TensorFillRandom<ElementInputA>(input.host_data(), (size_t)N * iH * iW *iC, 1, ElementInputA(1), ElementInputA(-1));
    TensorFillRandom<ElementInputB>(filter.host_data(), (size_t)oC * kH * kW * iC, 1, ElementInputB(1), ElementInputB(-1));
    TensorFillRandom<ElementOutput>(bias.host_data(), (size_t)oC, 1, ElementOutput(1), ElementOutput(-1));
    cutlass::reference::host::TensorFill(output.host_view());
    cutlass::reference::host::TensorFill(output_ref.host_view());
    input.sync_device();
    filter.sync_device();
    bias.sync_device();
    output.sync_device();
    output_ref.sync_device();
    
    hipStream_t stream = nullptr;
    ConvBias conv_bias(input.device_data(), filter.device_data(), bias.device_data(), output.device_data(),
            N, iC, iH, iW, oC, kH, kW, oH, oW, strideH, strideW, paddingH, paddingW, stream);
    conv_bias.forward();

    Conv conv(input.device_data(), filter.device_data(), output_ref.device_data(),
            N, iC, iH, iW, oC, kH, kW, oH, oW, strideH, strideW, paddingH, paddingW, stream);
    conv.forward();

    hipDeviceSynchronize();
    output.sync_host();
    output_ref.sync_host();

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < oH; j++) {
            for (int k = 0; k < oW; k++) {
                for (int l = 0; l < oC; l++) {
                    output_ref.at({i, j, k, l}) = std::max(ElementOutput(0), output_ref.at({i, j, k, l}) + bias.at({0, 0, 0, l}));
                }
            }
        }
    }

    bool passed = TensorEquals<ElementOutput>(output.host_data(), output_ref.host_data(), (size_t)N * oC * oH * oW);
    if (!passed) {
        printf("ERROR - results miscompared.\n");
    }

    return 0; 
}