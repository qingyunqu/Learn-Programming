#include "cudnn_convbias.h"
#include "../helper.h"
#include "../check.h"

#include "cutlass/tensor_coord.h"
#include "cutlass/layout/layout.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"

#include "hip/hip_fp16.h"

using ElementInputA = float;
using ElementInputB = float;
using ElementOutput = float;

using LayoutInputA = cutlass::layout::TensorNHWC;
using LayoutInputB = cutlass::layout::TensorNHWC;
using LayoutOutput = cutlass::layout::TensorNHWC;

int main() {
    int N = 8;
    int iC = 128;  // C
    int iH = 224;
    int iW = 224;
    int oC = 128;  // K
    int kH = 3;    // R
    int kW = 3;    // S
    int strideH = 1;
    int strideW = 1;
    int paddingH = 1;  // int paddingH0 = 1; int paddingH1 = 1;
    int paddingW = 1;  // int paddingW0 = 1; int paddingW1 = 1;
    int oH = (iH + 2 * paddingH - kH) / strideH + 1;  // P
    int oW = (iW + 2 * paddingW - kW) / strideW + 1;  // Q

    cutlass::Tensor4DCoord input_size(N, iH, iW, iC);
    cutlass::Tensor4DCoord filter_size(oC, kH, kW, iC);
    cutlass::Tensor4DCoord bias_size(1, 1, 1, oC);
    cutlass::Tensor4DCoord output_size(N, oH, oW, oC);
    cutlass::HostTensor<ElementInputA, LayoutInputA> input(input_size);
    cutlass::HostTensor<ElementInputB, LayoutInputB> filter(filter_size);
    cutlass::HostTensor<ElementOutput, LayoutOutput> bias(bias_size);
    cutlass::HostTensor<ElementOutput, LayoutOutput> output(output_size);

    TensorFillRandom<ElementInputA>(input.host_data(), (size_t)N * iH * iW *iC, 1, ElementInputA(1), ElementInputA(-1));
    TensorFillRandom<ElementInputB>(filter.host_data(), (size_t)oC * kH * kW * iC, 1, ElementInputB(1), ElementInputB(-1));
    TensorFillRandom<ElementOutput>(bias.host_data(), (size_t)oC, 1, ElementOutput(1), ElementOutput(-1));
    cutlass::reference::host::TensorFill(output.host_view());
    input.sync_device();
    filter.sync_device();
    bias.sync_device();
    output.sync_device();

    hipStream_t stream = nullptr;
    ConvBias<ElementInputA, ElementInputB, ElementOutput> conv_bias(input.device_data(), filter.device_data(), bias.device_data(), output.device_data(),
            N, iC, iH, iW, oC, kH, kW, oH, oW, strideH, strideW, paddingH, paddingW, stream);
    // warms up
    for (int i = 0; i < 4; i++) {
        conv_bias.forward();
    }
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    constexpr int runs = 10;
    for (int i = 0; i < runs; i++) {
        conv_bias.forward();
    }
    hipEventRecord(stop);
    hipDeviceSynchronize();
    float time;
    hipEventElapsedTime(&time, start, stop);
    std::cout << "cudnn time: " << time / runs << " ms\n";

    return 0;
}