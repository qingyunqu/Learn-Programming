#include <cassert>
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

template <typename T>
__global__ void reduce_sum_version_0(const T* src, T* dst, int M, int N) {
    const T* src_ptr = src + blockIdx.x * N;
    dst[blockIdx.x] = static_cast<T>(0);
    for (int i = 0; i < N; i++) {
        dst[blockIdx.x] += src_ptr[i];
    }
}

#define BLOCK_SIZE 1024
template <typename T>
__global__ void reduce_sum_version_1(const T* src, T* dst, T* buffer, int M,
                                     int N) {
    const T* src_ptr = src + blockIdx.x * N + threadIdx.x * BLOCK_SIZE;
    T* buffer_ptr = buffer + blockIdx.x * blockDim.x + threadIdx.x;
    *buffer_ptr = static_cast<T>(0);
    for (int i = 0; i < BLOCK_SIZE && threadIdx.x * BLOCK_SIZE + i < N; i++) {
        *buffer_ptr += src_ptr[i];
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        dst[blockIdx.x] = static_cast<T>(0);
        buffer_ptr = buffer + blockIdx.x * blockDim.x;
        for (int i = 0; i < blockDim.x; i++) {
            dst[blockIdx.x] += buffer_ptr[i];
        }
    }
}

template <typename T>
__global__ void reduce_sum_version_2(const T* src, T* dst, int M, int N) {
    extern __shared__ T buffer[];
    const T* src_ptr = src + blockIdx.x * N + threadIdx.x * BLOCK_SIZE;
    buffer[threadIdx.x] = static_cast<T>(0);
    for (int i = 0; i < BLOCK_SIZE && threadIdx.x * BLOCK_SIZE + i < N; i++) {
        buffer[threadIdx.x] += src_ptr[i];
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        dst[blockIdx.x] = static_cast<T>(0);
        for (int i = 0; i < blockDim.x; i++) {
            dst[blockIdx.x] += buffer[i];
        }
    }
}

int* init_host_input(int M, int N) {
    int* host_input = (int*)malloc(sizeof(int) * M * N);
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            host_input[i * N + j] = rand() % 1000;
        }
    }
    return host_input;
}

void host_reduce_sum(int* input, int* output, int M, int N) {
    for (int i = 0; i < M; i++) {
        output[i] = 0;
        for (int j = 0; j < N; j++) {
            output[i] += input[i * N + j];
        }
    }
}

// reduce size: M * N -> M
int main(int argc, char* argv[]) {
    if (argc < 4) {
        printf("Usage: ./%s <M> <N> <kernel>\n", argv[0]);
        return 0;
    }

    int M = atoi(argv[1]);
    int N = atoi(argv[2]);
    int kernel_id = atoi(argv[3]);

    int* host_input = init_host_input(M, N);
    int* host_output = (int*)malloc(sizeof(int) * M);
    host_reduce_sum(host_input, host_output, M, N);

    int *device_input = nullptr, *device_output = nullptr;
    int* device_buffer = nullptr;
    hipMalloc(&device_input, sizeof(int) * M * N);
    hipMalloc(&device_output, sizeof(int) * M);
    hipMalloc(&device_buffer, sizeof(int) * M * N);
    hipMemcpy(device_input, host_input, sizeof(int) * M * N,
               hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    int times = 1000;
    hipEventRecord(start, (hipStream_t)0);
    for (int i = 0; i < times; i++) {
        switch (kernel_id) {
            case 0:
                reduce_sum_version_0<int>
                        <<<M, 1>>>(device_input, device_output, M, N);
                break;
            case 1:
                reduce_sum_version_1<int>
                        <<<M, (N + BLOCK_SIZE - 1) / BLOCK_SIZE>>>(
                                device_input, device_output, device_buffer, M,
                                N);
                break;
            case 2: {
                int threads = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
                reduce_sum_version_2<int>
                        <<<M, threads, sizeof(int) * threads>>>(
                                device_input, device_output, M, N);
                break;
            }
            default:
                break;
        }
    }
    hipEventRecord(end, (hipStream_t)0);
    hipEventSynchronize(end);
    float time = 0.f;
    hipEventElapsedTime(&time, start, end);
    printf("M %d N %d kernel %d: average time %.2fms\n", M, N, kernel_id,
           time / times);

    int* host_device_output = (int*)malloc(sizeof(int) * M);
    hipMemcpy(host_device_output, device_output, sizeof(int) * M,
               hipMemcpyDeviceToHost);
    for (int i = 0; i < M; i++) {
        if (host_device_output[i] != host_output[i]) {
            printf("index: %d, not equal %d <-> %d\n", host_output[i],
                   host_device_output[i]);
            break;
        }
    }
    free(host_device_output);

    hipEventDestroy(start);
    hipEventDestroy(end);
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_buffer);
    free(host_input);
    free(host_output);
    return 0;
}
